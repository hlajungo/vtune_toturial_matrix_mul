#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

float tdiff(struct timeval *start, struct timeval *end)
{
  return (end->tv_sec - start->tv_sec) + 
    1e-6 * (end->tv_usec - start->tv_usec);
}


#define N 4096  // 矩陣大小

// CUDA 核心函數
__global__ void matrixMulKernel(double *A, double *B, double *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        double sum = 0;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    // 分配主機端記憶體
    double *h_A = (double *)malloc(N * N * sizeof(double));
    double *h_B = (double *)malloc(N * N * sizeof(double));
    double *h_C = (double *)malloc(N * N * sizeof(double));

    // 初始化矩陣
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = (double)rand() / RAND_MAX;
        h_B[i] = (double)rand() / RAND_MAX;
    }

  struct timeval start2, end2;
  gettimeofday(&start2, NULL);


    // 分配裝置端記憶體
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * N * sizeof(double));
    hipMalloc((void **)&d_B, N * N * sizeof(double));
    hipMalloc((void **)&d_C, N * N * sizeof(double));

    // 將資料從主機傳輸到裝置
    hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(double), hipMemcpyHostToDevice);

    // 定義 CUDA 核心的執行配置
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 計時開始
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    // 執行 CUDA 核心
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 計時結束
    hipEventRecord(end);
    hipEventSynchronize(end);

    // 計算執行時間
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    printf("Execution time: %0.6f ms\n", milliseconds);

    // 將結果從裝置傳回主機
    hipMemcpy(h_C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);

    // 釋放記憶體
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

  gettimeofday(&end2, NULL);
  printf("%0.6f\n", tdiff(&start2, &end2));


    return 0;
}

